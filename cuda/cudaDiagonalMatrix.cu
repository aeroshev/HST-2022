#include "hip/hip_runtime.h"
// C++ Libs
#include <iostream>
// CUDA libs
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;


#define CUDA_CHECK(call)                                                        \
    if ((call) != hipSuccess) {                                                \
        hipError_t err = hipGetLastError();                                   \
        cerr << "CUDA error calling \"" #call "\", code is " << err << "\n";    \
        exit(1);                                                                \
    }


void printMatrix(float *matrix, size_t N) {
    for (size_t i = 0; i < N; i++) {
        for (size_t j = 0; j < N; j++) {
            cout << matrix[i * N + j] << ' ';
        }
        cout << '\n';
    }
}


__global__ void matrixDiagonal(float *A, float *sum, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int column = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < column) {
        *sum += A[row * N + column];
    }

 }


int main(int argc, char *argv[]) {
    int N = 8;
    size_t matrixSize = sizeof(float) * N * N;
    size_t floatSize = sizeof(float);

    // Allocate matrix on CPU
    float *h_matrix_A = (float *)malloc(matrixSize);
    for (size_t i = 0; i < N * N; i ++) {
        // h_matrix_A[i] = rand() / (float)RAND_MAX;
        h_matrix_A[i] = 1;  // debug
    }
    printMatrix(h_matrix_A, N);

    float *h_sum = (float *)calloc(1, floatSize);

    // Allocate memory to GPU Device
    float *d_matrix_A = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_matrix_A, matrixSize));

    float *d_sum = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_sum, floatSize));

    // Copy data from CPU to GPU
    CUDA_CHECK(hipMemcpy(d_matrix_A, h_matrix_A, matrixSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sum, h_sum, floatSize, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 threadsPerBlock(N / 2, N / 2);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    matrixDiagonal<<<numBlocks, threadsPerBlock>>>(d_matrix_A, d_sum, N);

    // Copy result from GPU to CPU
    CUDA_CHECK(hipMemcpy(h_sum, d_sum, floatSize, hipMemcpyDeviceToHost));

    cout << "Result of sum:" << *h_sum << "\n";

    // Free memory
    CUDA_CHECK(hipFree(d_matrix_A));
    CUDA_CHECK(hipFree(d_sum));

    free(h_matrix_A);
    free(h_sum);

    return 0;
}
