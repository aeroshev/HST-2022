#include "hip/hip_runtime.h"
// C++ Libs
#include <iostream>
// CUDA libs
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;


#define CUDA_CHECK(call)                                                        \
    if ((call) != hipSuccess) {                                                \
        hipError_t err = hipGetLastError();                                   \
        cerr << "CUDA error calling \"" #call "\", code is " << err << "\n";    \
        exit(1);                                                                \
    }


void printMatrix(float *matrix, size_t N) {
    for (size_t i = 0; i < N; i++) {
        for (size_t j = 0; j < N; j++) {
            cout << matrix[i * N + j] << ' ';
        }
        cout << '\n';
    }
}


__global__ void matrixDiagonal(float *A, float *sum, int N) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < N; i++) {
        if (row < i) {
            sum[row] += A[row * N + i];
        }  
    }
 }


int main(int argc, char *argv[]) {
    int N = 8;
    size_t matrixSize = sizeof(float) * N * N;
    size_t vectorSize = sizeof(float) * N;

    // Allocate matrix on CPU
    float *h_matrix_A = (float *)malloc(matrixSize);
    for (size_t i = 0; i < N * N; i++) {
        // h_matrix_A[i] = rand() / (float)RAND_MAX;
        h_matrix_A[i] = 1;  // debug
    }
    printMatrix(h_matrix_A, N);

    float *h_vector_sum = (float *)calloc(N, sizeof(float));

    // Allocate memory to GPU Device
    float *d_matrix_A = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_matrix_A, matrixSize));

    float *d_vector_sum = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_vector_sum, vectorSize));

    // Copy data from CPU to GPU
    CUDA_CHECK(hipMemcpy(d_matrix_A, h_matrix_A, matrixSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_vector_sum, h_vector_sum, vectorSize, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 threadsPerBlock(N / 2);
    dim3 numBlocks(N / threadsPerBlock.x);
    matrixDiagonal<<<numBlocks, threadsPerBlock>>>(d_matrix_A, d_vector_sum, N);

    // Copy result from GPU to CPU
    CUDA_CHECK(hipMemcpy(h_vector_sum, d_vector_sum, vectorSize, hipMemcpyDeviceToHost));

    float sum = 0;
    for (int i = 0; i < N; i++) {
        sum += h_vector_sum[i];
    }

    cout << "Result of sum:" << sum << "\n";

    // Free memory
    CUDA_CHECK(hipFree(d_matrix_A));
    CUDA_CHECK(hipFree(d_vector_sum));

    free(h_matrix_A);
    free(h_vector_sum);

    return 0;
}
